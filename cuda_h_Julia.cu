#include "hip/hip_runtime.h"
#include<iostream>
#include "lodepng.h"
#include<vector>
#include<fstream>
//#include<math.h>
#include<sys/time.h>


#define T 1024 //number of threads/block(32,32)
#define N 64 //number of blocks per grid (256,256)
//Encode from raw pixels to disk with a single function call
//he image argument has width * height RGBA pixels or width * height * 4 bytes
void encodeImage(const char* filename, unsigned char const* image, unsigned width, unsigned height)
{
	//Encode the image
	unsigned error = lodepng::encode(filename, image, width, height);
	//if there's an error, display it
	if (error) std::cout << "encoder error " << error << ": " << lodepng_error_text(error) << std::endl;
}

double getSeconds()
{
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return ((double)tp.tv_sec + (double)tp.tv_usec * 1e-6);
}

__global__ void evalJulia(float h,

			  unsigned int max_iteration,

			  float pixel_limit,

			  float c_real,

			  float c_img,

			  unsigned char* colourBit,

			  long img_size){

	
	//printf("%f %d %d %f %f %d\n", h, max_iteration, pixel_limit, c_real, c_img, img_size);
	long id = threadIdx.x + blockIdx.x*blockDim.x;
	long x_index = id%img_size;
	long y_index = id/img_size;
	//long y_index = threadIdx.y + blockIdx.y*blockDim.y;
	float real = -2.0 + h * (id%img_size);
	float img = -2.0 + h * (id/img_size);
	float mod = real*real + img*img;
	float temp=0;
	int iter = 0;
	printf("%d %d\n", id, int(id/img_size));
	while ((mod <= (pixel_limit*pixel_limit)) && (iter < max_iteration))
		{
		//printf("real img mod %f %f %f %d\n", real, img, mod, iter);
		temp = (real*real) - (img*img) + 0.0000;
		img = 2.0*real*img - 0.800;
		real = temp;
		mod = (real * real) + (img * img);
		//printf("real img mod %f %f %f %d\n", real, img, mod, iter);
		iter = iter + 1;
		}
	//printf("inside the loop %d %d %f %f %d\n", x_index, y_index, real, img, iter);	
		//printf("pixel value %d\n", int((iter/200.0)*255));
	//update colour
                        colourBit[4 *(img_size)*y_index + 4 * x_index + 3] = 255;
                        colourBit[4 *(img_size)*y_index + 4 * x_index + 2] = 0;
                        colourBit[4 *(img_size)*y_index + 4 * x_index + 1] = 0;//(unsigned char)iter;
                        colourBit[4 *(img_size)*y_index + 4 * x_index + 0] = int(iter/200.0)*255; 
	
		
}




int main()
{
	long img_size = 2048; // Image size(64x64)
		
	//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
	//std::cout<<"//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++"<<std::endl;
	float spacing = 4.0 / (double)img_size; //spacing is length/image size
	//std::cout<<"spacing "<<spacing<<std::endl;	
	float pixel_limit = 20;
	float c_real = 0.0;	//constant complex real
	float c_img = 0.8; 	//constant complex imaginary
	//std::cout << "before kernel cal"<<std::endl;
	unsigned int iteration_limit = 100; // maximum number of iterations
	unsigned char*  colourBit = new unsigned char[img_size*img_size * 4];
	unsigned char* d_colourBit;
	hipMalloc((void**)&d_colourBit, (img_size*img_size*4*sizeof(unsigned char)));
	double wcTimeStart= 0.0, wcTimeEnd=0.0;
	//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
	//computuation begins here:
	wcTimeStart = getSeconds(); //Start time
	//dim3 gridDim(128,128); //(64,64)
	//dim3 blockDim(16,16); //16 implicitly considered (number of threads in x and y directions) (16,16)
	long threads_Block = 1024;
	long blocks = (2048*2048)/threads_Block;
	//long len = 2048;
	evalJulia<<<blocks, threads_Block>>>(spacing, iteration_limit, pixel_limit, c_real, c_img, d_colourBit, img_size);
	hipError_t errSync  = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess) 
 		 printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
  	printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
	hipMemcpy(colourBit,d_colourBit, (img_size*img_size*sizeof(unsigned char)),hipMemcpyDeviceToHost);
	wcTimeEnd = getSeconds(); //End time
	std::cout << "Done with operations, begin image encoding!" << std::endl;
	std::cout << "Time Taken for computation: " << wcTimeEnd-wcTimeStart << " sec" << std::endl;
	encodeImage("JuliaCPU.png", colourBit, img_size, img_size);
	std::cout << "The image has been generated and is named as JuliaCPU.png" << std::endl;
	std::cout << "Time Taken for image encoding: " << (wcTimeEnd-wcTimeStart)*1e3 << " milli-sec" << std::endl;
	hipFree(d_colourBit);
	delete(colourBit);
	return 0;
}


