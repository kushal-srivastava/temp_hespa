#include "hip/hip_runtime.h"
#include<iostream>
#include "lodepng.h"
#include<vector>
#include<fstream>
//#include<math.h>
#include<sys/time.h>


#define T 1024 //number of threads/block(32,32)
#define N 64 //number of blocks per grid (256,256)
//Encode from raw pixels to disk with a single function call
//he image argument has width * height RGBA pixels or width * height * 4 bytes
void encodeImage(const char* filename, unsigned char const* image, unsigned width, unsigned height)
{
	//Encode the image
	unsigned error = lodepng::encode(filename, image, width, height);
	//if there's an error, display it
	if (error) std::cout << "encoder error " << error << ": " << lodepng_error_text(error) << std::endl;
}

double getSeconds()
{
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return ((double)tp.tv_sec + (double)tp.tv_usec * 1e-6);
}
//__global__ void evalJulia(double*, int*, int, int, double, double, unsigned char*, const unsigned int);
__global__ void evalJulia(const double h,

			  int max_iteration,

			  int pixel_limit,

			  double c_real,

			  double c_img,

			  unsigned char* colourBit,

			  const unsigned int img_size){

	

	int x_index = threadIdx.x + blockIdx.x*blockDim.x;

	int y_index = threadIdx.y + blockIdx.y*blockDim.y;
//	printf("%f
	//int index = x_index + (T/32)*y_index;
	double real = -2.0 + h * (double)x_index;
	double img = -2.0 + h * (double)y_index;
	//printf("%d %d \n ", x_index, y_index);
	double mod = real*real + img*img;
	double temp;
	int iter = 0;
	while (mod <= (pixel_limit*pixel_limit) && iter < max_iteration){
		temp = real*real - img*img + c_real;
		img = 2*real*img + c_img;
		real = temp;
		mod = real * real + img * img;
		iter++;}
	//__syncthreads();
	printf("%d\n", y_index);
	//update colour/*
                        /*colourBit[4 *(img_size)*y_index + 4 * x_index + 3] = 255;
                        colourBit[4 *(img_size)*y_index + 4 * x_index + 2] = 0;
                        colourBit[4 *(img_size)*y_index + 4 * x_index + 1] = (unsigned char)iter;
                        colourBit[4 *(img_size)*y_index + 4 * x_index + 0] = ((double)iter/200.0)*255;*/
	//d_iteration[index] = iter;
	__syncthreads();
}




int main()
{
	const unsigned int img_size = 2048; // Image size
	//double *range = new double[img_size]; //Stores all the values between range -2 to 2 with spacing
	//double *d_range;
	//hipMalloc((void**)&d_range, img_size*sizeof(double));
	
	//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
	const double spacing = 4.0 / (double)img_size; //spacing is length/image size
	const double & h = spacing;	//spacing alias
		std::cout<<h<<std::endl;
	//convert the image size to be in the range of -2 to 2
	/*for (unsigned int i = 0; i < img_size; ++i)
	{
		range[i] = -2.0 + (double)(i)*h;
	}*/
	//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
	//int tmp = img_size*sizeof(double);
	//hipMemcpy(range, d_range, (img_size*sizeof(double)), hipMemcpyHostToDevice);
	int pixel_limit = 12;
	double c_real = -0.8;	//constant complex real
	double c_img = 0.2; 	//constant complex imaginary
	//int* iteration = new int[img_size*img_size]; //iterations done per pixel
	//int* d_iteration;
	std::cout << "before kernel cal"<<std::endl;
	//hipMalloc((void**)&d_iteration, (img_size*img_size*sizeof(int)));
	int iteration_limit = 50; // maximum number of iterations
	unsigned char*  colourBit = new unsigned char[img_size*img_size * 4];
	double wcTimeStart= 0.0, wcTimeEnd=0.0;
	//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
	//computuation begins here:
	wcTimeStart = getSeconds(); //Start time
	dim3 gridDim(128,128); //(64,64)
	dim3 blockDim(16,16); //16 implicitly considered (number of threads in x and y directions) (16,16)
	evalJulia<<<gridDim, blockDim>>>(h, iteration_limit, pixel_limit, c_real, c_img, colourBit, img_size);
	hipError_t errSync  = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess) 
 		 printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
  	printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
//	__syncthreads();
	//hipMemcpy(d_iteration,iteration, (img_size*img_size*sizeof(int)),hipMemcpyDeviceToHost);
	wcTimeEnd = getSeconds(); //End time
	std::cout << "Done with operations, begin image encoding!" << std::endl;
	std::cout << "Time Taken for computation: " << wcTimeEnd-wcTimeStart << " sec" << std::endl;
	encodeImage("JuliaCPU.png", colourBit, img_size, img_size);
	std::cout << "The image has been generated and is named as JuliaCPU.png" << std::endl;
	std::cout << "Time Taken for image encoding: " << (wcTimeEnd-wcTimeStart)*1e3 << " milli-sec" << std::endl;
	
	//hipFree(d_iteration);
	//hipFree(d_range);
	//delete(range);
	//delete(iteration);
	delete(colourBit);
	return 0;
}


