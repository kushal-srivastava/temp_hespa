#include "hip/hip_runtime.h"
#include<iostream>
#include "lodepng.h"
#include<vector>
#include<fstream>
//#include<math.h>
#include<sys/time.h>


#define T 1024 //number of threads/block(32,32)
#define N 64 //number of blocks per grid (256,256)
//Encode from raw pixels to disk with a single function call
//he image argument has width * height RGBA pixels or width * height * 4 bytes
void encodeImage(const char* filename, unsigned char const* image, unsigned width, unsigned height)
{
	//Encode the image
	unsigned error = lodepng::encode(filename, image, width, height);
	//if there's an error, display it
	if (error) std::cout << "encoder error " << error << ": " << lodepng_error_text(error) << std::endl;
}

double getSeconds()
{
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return ((double)tp.tv_sec + (double)tp.tv_usec * 1e-6);
}
//__global__ void evalJulia(double*, int*, int, int, double, double, unsigned char*, const unsigned int);
__global__ void evalJulia(double h,

			  int max_iteration,

			  int pixel_limit,

			  double c_real,

			  double c_img,

			  unsigned char* colourBit,

			  unsigned int img_size){

	

	int x_index = threadIdx.x + blockIdx.x*blockDim.x;

	int y_index = threadIdx.y + blockIdx.y*blockDim.y;
	double real = -2.0 + h * (double)x_index;
	double img = -2.0 + h * (double)y_index;
	double mod = real*real + img*img;
	double temp;
	int iter = 0;
	while (mod <= (pixel_limit*pixel_limit) && iter < max_iteration){
		temp = real*real - img*img + c_real;
		img = 2*real*img + c_img;
		real = temp;
		mod = real * real + img * img;
		iter++;}
		
	//update colour
                        colourBit[4 *(img_size)*y_index + 4 * x_index + 3] = 255;
                        colourBit[4 *(img_size)*y_index + 4 * x_index + 2] = 0;
                        colourBit[4 *(img_size)*y_index + 4 * x_index + 1] = (unsigned char)iter;
                        colourBit[4 *(img_size)*y_index + 4 * x_index + 0] = ((double)iter/200.0)*255; 
	
}




int main()
{
	unsigned int img_size = 2048; // Image size
		
	//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
	double spacing = 4.0 / (double)img_size; //spacing is length/image size
	//const double & h = spacing;	//spacing alias
//		std::cout<<h<<std::endl;
	//convert the image size to be in the range of -2 to 2
	/*for (unsigned int i = 0; i < img_size; ++i)
	{
		range[i] = -2.0 + (double)(i)*h;
	}*/
	//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
	//int tmp = img_size*sizeof(double);
	//hipMemcpy(range, d_range, (img_size*sizeof(double)), hipMemcpyHostToDevice);
	int pixel_limit = 12;
	double c_real = -0.8;	//constant complex real
	double c_img = 0.2; 	//constant complex imaginary
	//int* iteration = new int[img_size*img_size]; //iterations done per pixel
	//int* d_iteration;
	std::cout << "before kernel cal"<<std::endl;
	//hipMalloc((void**)&d_iteration, (img_size*img_size*sizeof(int)));
	int iteration_limit = 50; // maximum number of iterations
	unsigned char*  colourBit = new unsigned char[img_size*img_size * 4];
	unsigned char* d_colourBit;
	hipMalloc((void**)&d_colourBit, (img_size*img_size*4*sizeof(unsigned char)));
	double wcTimeStart= 0.0, wcTimeEnd=0.0;
	//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
	//computuation begins here:
	wcTimeStart = getSeconds(); //Start time
	dim3 gridDim(64,64); //(64,64)
	dim3 blockDim(32,32); //16 implicitly considered (number of threads in x and y directions) (16,16)
	evalJulia<<<gridDim, blockDim>>>(spacing, iteration_limit, pixel_limit, c_real, c_img, d_colourBit, img_size);
	hipError_t errSync  = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess) 
 		 printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
  	printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
//	__syncthreads();
	hipMemcpy(d_colourBit,colourBit, (img_size*img_size*sizeof(unsigned char)),hipMemcpyDeviceToHost);
	wcTimeEnd = getSeconds(); //End time
	std::cout << "Done with operations, begin image encoding!" << std::endl;
	std::cout << "Time Taken for computation: " << wcTimeEnd-wcTimeStart << " sec" << std::endl;
	encodeImage("JuliaCPU.png", colourBit, img_size, img_size);
	std::cout << "The image has been generated and is named as JuliaCPU.png" << std::endl;
	std::cout << "Time Taken for image encoding: " << (wcTimeEnd-wcTimeStart)*1e3 << " milli-sec" << std::endl;
	
	//hipFree(d_iteration);
	//hipFree(d_range);
	//delete(range);
	//delete(iteration);
	delete(colourBit);
	return 0;
}


