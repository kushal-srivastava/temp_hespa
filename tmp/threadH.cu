
#include <hip/hip_runtime.h>
#include<iostream>
//#include<stdio.h>
//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
    __global__ void evalJulia(int *d_pixel, 
                              int *d_temp){
        
				int x_index = threadIdx.x + blockIdx.x*blockDim.x;
				int y_index = threadIdx.y + blockIdx.y*blockDim.y;
				int tmp = x_index + 2*y_index;
				d_temp[tmp] = d_pixel[tmp];
		   		 			
			}

//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
#define N 64
int main()
{

	int *d_pixel;
	int *d_temp;

	int size = N * sizeof(int);
    //+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
	int *temp = new int[N];
    	int *h_temp = new int[N];
    	for (int y=0;y<8;y++)
        	for(int x=0;x<8;x++)
        	{
            		temp[x + 8*y] = x + 8*y;
            		std::cout<<temp[x+8*y]<<std::endl;
        	}
	//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
	std::cout<<"test begins"<<std::endl;
    	dim3 gridDim(4,4);
    	dim3 blockDim(2,2);
    	hipMalloc((void**)&d_pixel, size);
    	hipMalloc((void**)&d_temp, size);
    	hipMemcpy(temp, d_pixel, size, hipMemcpyHostToDevice);
    	//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
    	evalJulia<<<gridDim,blockDim>>>(d_pixel, d_temp);
    	hipMemcpy(h_temp, d_temp, size, hipMemcpyDeviceToHost);
    	//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++    
   	for (int y=0;y<8;y++)
        	for(int x=0;x<8;x++)
        	{
			std::cout<<temp[x+8*y]<<std::endl;}
    	std::cout<<"last kernel thread printed"<<std::endl;
    	hipFree(d_pixel);
    	hipFree(d_temp);
    	delete(h_temp);
    	delete(temp);
	return 0;
}
