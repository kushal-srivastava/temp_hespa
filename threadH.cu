
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
    __global__ void evalJulia(double *d_pixel, 
                              double *d_temp){
        
				int x_index = threadIdx.x + 2*threadIdx.y + 4*(blockIdx.x + blockIdx.y);
				d_temp[x_index] = d_pixel[x_index];
		   		 			
			}

//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
int main()
{

	double *d_pixel;
	double *d_temp;
	const int size = 16*16;
    //+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
    double *temp = new double(size);
    double *h_temp = new double(size);
    for (int y=0;y<16;y++)
        for(int x=0;x<16;x++)
        {
            temp[x + 16*y] = x + 16*y;
            std::cout<<temp[x+16*y]<<std::endl;
        }
	//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
	std::cout<<"test begins"<<std::endl;
    dim3 threadsPerBlock(2,2);
    dim3 numBlocks(8,8);
    hipMalloc((void**)&d_pixel, size);
    hipMalloc((void**)&d_temp, size);
    //cudaMemcpy(temp, d_pixel, size, cudaMemcpyHostToDevice);
    //+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
    //evalJulia<<<numBlocks,threadsPerBlock>>>(d_pixel, d_temp);
    //cudaMemcpy(h_temp, d_temp, size, cudaMemcpyDeviceToHost);
    //++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++    
/*    for (int y=0;y<2048;y++)
        for(int x=0;x<2048;x++)
        {std::cout<<temp[x+2048*y]<<std::endl;}*/
    std::cout<<"last kernel thread printed"<<std::endl;
    hipFree(d_pixel);
    hipFree(d_temp);
    delete(h_temp);
    delete(temp);
	return 0;
}
