
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>

__global__ void kern(void){
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
//	printf("Dim %d %d \n", blockDim.x, blockDim.y);	
	printf("%d %d %d\n", x,y, (x + 8*y));
//	__syncthreads();
	printf("Id%d %d %d\n", blockIdx.x, blockIdx.y, (x + 8*y));
//	printf("%d \n", x + 8*y);
}

int main(){
	dim3 gridDim(2,2);
	dim3 blockDim(4,4);
	kern<<<gridDim, blockDim>>>();
	hipError_t errSync  = hipGetLastError();
hipError_t errAsync = hipDeviceSynchronize();
if (errSync != hipSuccess) 
  printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
if (errAsync != hipSuccess)
  printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
	return 0;
}
